#include "hip/hip_runtime.h"
/* /////////////// DISCLAIMER/////////////////////////////////
   This software is provided by the author and
   contributors ``as is'' and any express or implied
   warranties, including, but not limited to, the
   implied warranties of merchantability and
   fitness for a particular purpose are dis-
   claimed. In no event shall the author or con-
   tributors be liable for any direct, indirect,
   incidental, special, exemplary, or consequen-
   tial damages (including, but not limited to,
   procurement of substitute goods or services;
   loss of use, data, or profits; or business
   interruption) however caused and on any
   theory of liability, whether in contract,
   strict liability, or tort (including negligence
   or otherwise) arising in any way out of the use
   of this software, even if advised of the poss-
   ibility of such damage.
//////////////////////////////////////////////////////*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "F5crypt.h"

#include "sha1.cu"
#include "hash_sha1.cu"

__device__ char deZigZag[] = {  0,  1,  5,  6, 14, 15, 27, 28,
                     2,  4,  7, 13, 16, 26, 29, 42,
                     3,  8, 12, 17, 25, 30, 41, 43,
                     9, 11, 18, 24, 31, 40, 44, 53,
                    10, 19, 23, 32, 39, 45, 52, 54,
                    20, 22, 33, 38, 46, 51, 55, 60,
                    21, 34, 37, 47, 50, 56, 59, 61,
                    35, 36, 48, 49, 57, 58, 62, 63};

__device__ char pk_sentinal[] = "----* PK v 1.0 REQUIRES PASSWORD ----*";


__device__ void F5gen_next_batch(f5_rand_state *st) {
	// From SecureRandom.getEngineNextBytes
	// Conviently, the 20 state bytes get assembled in little endin order into 5 integers
	SHA1(st->output, st->state, 20);

	// From SecureRandom.updateState
	st->last = 1;
	st->v = 0;
	st->t = 0;
	st->zf = 0;  //using as bool

	for (int j = 0; j < 20; j++)
	{
		st->v = (int)st->state[j] + (int)st->output[j] + st->last;
		st->t = (uint8_t)st->v;   //lower 8 bits only
		st->zf = st->zf | (st->state[j] != st->t);
		st->state[j] = st->t;
		st->last = st->v >> 8;
	}

	if (!st->zf) st->state[0]++;  // make sure at least one bit changes

	// reset output position
	st->output_pos = 0;
}

__device__ void F5gen_rand_seed(char *seed, int seed_len, f5_rand_state *st)
{
	/*
	ALT_SHA1(st->state, seed, seed_len);

	printf("alt sha1 ");
	for (int offset = 0; offset < 20; offset++) {
		printf("%02x", st->state[offset] & 0xff);
	}
	printf("\n");
	*/
	SHA1(st->state, seed, seed_len);
	/*
	printf("sha1 ");
	for (int offset = 0; offset < 20; offset++) {
		printf("%02x", st->state[offset] & 0xff);
	}
	printf("\n");
	*/
	F5gen_next_batch(st);
}

__device__ int F5gen_rand_next(f5_rand_state *st)
{
	int rand = st->output[st->output_pos++];
	// Generate in blocks of 5 ints
	if (st->output_pos >= 20) {
		F5gen_next_batch(st);
	}
	return rand;
}

__device__ void F5permutation(f5_rand_state *st, int *shuffled, int size)
{                                     /*modifies*/
	// finishes what F5Random does
	// then builds the permutation table
	int random_index;
	int tmp;
	int max_random;

	for (int i = 0; i < size; i++) {
		shuffled[i] = i;
	}
	max_random = size;
	for (int i = 0; i < size; i++)
	{
		// F5Random.getNextValue
		random_index = F5gen_rand_next(st) | F5gen_rand_next(st) << 8 | F5gen_rand_next(st) << 16 | F5gen_rand_next(st) << 24;
		random_index %= max_random;
		if (random_index < 0) {
			random_index += max_random;
		}

		max_random--;

		tmp = shuffled[random_index];
		shuffled[random_index] = shuffled[max_random];
		shuffled[max_random] = tmp;
	}
	
#ifdef DEBUG
	FILE *debug_dump;
	debug_dump = fopen("permutation_dump.dat", "wb");
	if (debug_dump) fwrite(shuffled, 4, size, debug_dump);
	fclose(debug_dump);
#endif
}

__device__ int F5extract(short *coeff, int coeff_len, int* shuffled, f5_rand_state *st, int max_msg_length, char *message, int *message_len, int mode)
{
	int msg_pos = 0;
	// Taken strate from Extract.java
	char extractedByte = 0;
	int  availableExtractedBits = 0;
	int  extractedFileLength = 0;
	int  nBytesExtracted = 0;
	int  shuffledIndex = 0;
	int  extractedBit = 0;
	int  i;  //not only an iterator

	//extract the length of the hidden message
	for (i = 0; availableExtractedBits < 32; i++)
	{
		if (i >= coeff_len) {
			return 0;
		}
		shuffledIndex = shuffled[i];
		if (shuffledIndex % 64 == 0) continue; //Skip DCs
		shuffledIndex = shuffledIndex - (shuffledIndex % 64) + deZigZag[shuffledIndex % 64];
		if (coeff[shuffledIndex] == 0) continue; //Skip zeros
		if (coeff[shuffledIndex] > 0)
			extractedBit = coeff[shuffledIndex] & 1;        // if coefficent is greater than 0 then take LSB literally
		else
			extractedBit = 1 - (coeff[shuffledIndex] & 1);  // else, take the INVERSE of the LSB
		extractedFileLength |= extractedBit << availableExtractedBits++;
	}

	// remove pad
	extractedFileLength ^= F5gen_rand_next(st);
	extractedFileLength ^= F5gen_rand_next(st) << 8;
	extractedFileLength ^= F5gen_rand_next(st) << 16;
	extractedFileLength ^= F5gen_rand_next(st) << 24;
	int k = (extractedFileLength >> 24);
	k %= 32;
	int n = (1 << k) - 1;
	extractedFileLength &= 0x007fffff;

	// quick retun of message length is beyond  max reasonable size
	if ((extractedFileLength > max_msg_length) || (extractedFileLength < 16)) {
		return 0;
	}

	// pixel knot 
	if ((mode > 0) && (extractedFileLength < mode)) {
		return 0;
	}

	//	printf("file length %d\n", extractedFileLength);

	// Proceed with decode
	availableExtractedBits = 0;
	if (n > 0)
	{
		int startOfN = i;
		int hash;

		while (1)
		{
			// read places and calculate bits
			hash = 0;
			int code = 1;
			for (i = 0; code <= n; i++)
			{
				if (startOfN + i >= coeff_len) return 0;
				shuffledIndex = shuffled[startOfN + i];
				if (shuffledIndex % 64 == 0) continue; //skip DCs
				shuffledIndex = shuffledIndex - (shuffledIndex % 64) + deZigZag[shuffledIndex % 64];
				if (coeff[shuffledIndex] == 0) continue; //skip zeros
				if (coeff[shuffledIndex] > 0)
					extractedBit = coeff[shuffledIndex] & 1;
				else
					extractedBit = (1 - coeff[shuffledIndex]) & 1;
				if (extractedBit == 1) hash ^= code;
				code++;
			}
			startOfN += i;

			// write k bits bytewise
			for (i = 0; i < k; i++)
			{
				extractedByte |= ((hash >> i) & 1) << availableExtractedBits++;
				if (availableExtractedBits == 8)
				{
					// remove pad and save byte
					extractedByte ^= F5gen_rand_next(st);
					message[msg_pos++] = extractedByte;
					extractedByte = 0;
					availableExtractedBits = 0;
					nBytesExtracted++;

					// PixelKnot specific check
					if (mode > 0) {
						if (message[msg_pos - 1] != pk_sentinal[msg_pos - 1]) {
							// printf("wrong byte decoded\n");
							return 0;
						}
						if (msg_pos > mode) {
							printf("decoded enough matching bytes %d/%d\n", nBytesExtracted, extractedFileLength);
							*message_len = msg_pos;
							return 1;
						}
					}
				}
				// check for pending end of embedded data
				if (nBytesExtracted == extractedFileLength)
				{
					printf("extracted all bytes at %d\n", nBytesExtracted);
					*message_len = msg_pos;
					return nBytesExtracted > 1; //woohoo!
				}
			}
		}
	}
	else  //Default code used
	{
		for (/*see above*/; i < coeff_len; i++)
		{
			shuffledIndex = shuffled[i];
			if (shuffledIndex % 64 == 0) continue; //skip DCs
			shuffledIndex = shuffledIndex - (shuffledIndex % 64) + deZigZag[shuffledIndex % 64];
			if (coeff[shuffledIndex] == 0) continue; //skip zeros
			if (coeff[shuffledIndex] > 0)
				extractedBit = coeff[shuffledIndex] & 1;
			else
				extractedBit = 1 - (coeff[shuffledIndex] & 1);
			extractedByte |= extractedBit << availableExtractedBits++;
			if (availableExtractedBits == 8)
			{
				// remove pseudo random pad
				extractedByte ^= F5gen_rand_next(st);
				message[msg_pos++] = extractedByte;
				extractedByte = 0;
				availableExtractedBits = 0;
				nBytesExtracted++;

				// PixelKnot specific check
				if (mode > 0) {
					if (message[msg_pos - 1] != pk_sentinal[msg_pos - 1]) {
						// printf("wrong byte decoded\n");
						return 0;
					}
					if (msg_pos > mode) {
						printf("decoded enough matching bytes %d/%d\n", nBytesExtracted, extractedFileLength);
						*message_len = msg_pos;
						return 1;
					}
				}

				// extracted all requested bytes
				if (nBytesExtracted == extractedFileLength)
				{
					printf("extracted all bytes at %d\n", nBytesExtracted);
					*message_len = msg_pos;
					return nBytesExtracted > 0;
				}
			}
		}
	}

//	printf("decode failed at %d\n", nBytesExtracted);

	if (nBytesExtracted < extractedFileLength)
		return 0;

	return 0;  //shouldn't ever get to here
}

