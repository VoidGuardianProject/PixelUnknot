#include "hip/hip_runtime.h"
/* /////////////// DISCLAIMER/////////////////////////////////
   This software is provided by the author and
   contributors ``as is'' and any express or implied
   warranties, including, but not limited to, the
   implied warranties of merchantability and
   fitness for a particular purpose are dis-
   claimed. In no event shall the author or con-
   tributors be liable for any direct, indirect,
   incidental, special, exemplary, or consequen-
   tial damages (including, but not limited to,
   procurement of substitute goods or services;
   loss of use, data, or profits; or business
   interruption) however caused and on any
   theory of liability, whether in contract,
   strict liability, or tort (including negligence
   or otherwise) arising in any way out of the use
   of this software, even if advised of the poss-
   ibility of such damage.
//////////////////////////////////////////////////////*/

#include "hip/hip_runtime.h"
#include ""

#include <stddef.h>
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <string.h>
#include <time.h> 

#include "F5crypt.h"
#include "F5crypt.cu"

__device__ int device_strlen(char *str)
{
	if (!str) {
		return 0;
	}

	char *ptr = str;
	while (*str) {
		++str;
	}

	return str - ptr;
}

__device__ void device_decode(int idx, short *coeff, int coeff_len, int * perm_buffer, char * passwords, char * results, int max_msg_len, int max_pass, int max_decode)
{
	char * pass;
	int pass_len;
	f5_rand_state  rstate;
	int * perms = &(perm_buffer[idx * coeff_len]);
	pass = &(passwords[idx * max_pass]);
	pass_len = device_strlen(pass);
	// printf("%d perm: %p trying pass: \"%s\" len %d\n", idx, perms, pass, pass_len);
	F5gen_rand_seed(pass, pass_len, &rstate);
	/*
	printf("initial rands: ");
	for (int offset = 0; offset < 20; offset++) {
		printf("%02x", rstate->output[offset] & 0xff);
	}
	printf("\n");
	*/

	F5permutation(&rstate, perms, coeff_len);
	/*
	for (int j = 0; j < 10; j++) {
		printf("%d %s msg %p [%d] : %p = %d\n", idx, pass, perms, j, &(perms[j]), perms[j]);
	}
	*/

	char *msg = &(results[idx * (max_decode+1)]);
	int msg_len = 0;
	// 4. Attempt extraction
	// Return 1 on success, 0 on failure. On success message and message_len will be modified.
	int ret = F5extract(coeff, coeff_len, perms, &rstate, max_msg_len, msg, &msg_len, max_decode);
	msg[msg_len] = '\0';
	// 5. Write results if password found
	if (ret)
	{
		printf(">>>>> Password Hit: \"%s\" <<<<<\n", pass);
		printf("%s == %s\n", pass, msg);
	}
}

__global__ void decode(short *coeff, int coeff_len, int * perm_buffer, char * passwords, char * results, int max_msg_len, int max_pass, int max_decode)
{
	device_decode(blockIdx.x * blockDim.x + threadIdx.x, coeff, coeff_len, perm_buffer, passwords, results, max_msg_len, max_pass, max_decode);
}

int getline(char line[256], size_t *n, FILE *stream);
int load_coeff(char* filename, short** coeffptr, int* coeff_len, int *max_msg_len);
#define CUDA_ERR_CHECK 			if (cudaStatus != hipSuccess) { fprintf(stderr, "%d cuda returned error code %d : %s!\n", __LINE__, cudaStatus, hipGetErrorString(cudaStatus)); return 1; }


void usage()
{
	printf("--- BruthCrackF5CUDA ---\n");
	printf("Reads a provided coefficent dump from a JPEG file (TODO: jpeglib) and tests it\n");
	printf("against a seriesof passwords provided as a password file.\n");
	printf("\nUsage: brutecrackf5 filename [OPTION]...\n\nOptions:\n");
	printf(" --pass FILENAME Password list. Expected to be seperated by new-line charactors.\n");
	printf(" --gpu number\n");
	printf("                 Default: 0\n");
	printf(" --blocks count\n");
	printf("                 Default: 4\n");
	printf(" --threads count\n");
	printf("                 Default: 4\n");
	printf(" --max-pass max length of password\n");
	printf("                 Default: 8\n");
	printf(" --max-decode max number of bytes matching PixelKnot header to decode\n");
	printf("                 Default: 4\n");
	printf(" --suffix try all suffix of each password up to length\n");
	printf("                 Default: 0\n");
	printf(" --skip skip lines of password file\n");
	printf("                 Default: 0\n");

	exit(0);
}



int main(int argc, char** argv)
{
	char  * coeff_file = 0;
	char  * pass_file = 0;

	int n_blocks = 32;
	int n_threads = 32;

	int max_pass = 8;
	int max_decode = 4;
	int suffix_length = 0;
	int skip = 0;
	int gpu_id = 0;
	//Parse Args
	for (int i = 1; i < argc; i++)
	{
		if (strcmp(argv[i], "--pass") == 0 || strcmp(argv[i], "-p") == 0)
		{
			if (i + 1 == argc) usage();
			pass_file = argv[++i];
			continue;
		}
		if (strcmp(argv[i], "--gpu") == 0 || strcmp(argv[i], "-g") == 0)
		{
			if (i + 1 == argc) usage();
			skip = strtol(argv[++i], NULL, 10);
			continue;
		}
		if (strcmp(argv[i], "--skip") == 0 || strcmp(argv[i], "-sk") == 0)
		{
			if (i + 1 == argc) usage();
			skip = strtol(argv[++i], NULL, 10);
			continue;
		}
		if (strcmp(argv[i], "--blocks") == 0 || strcmp(argv[i], "-b") == 0)
		{
			if (i + 1 == argc) usage();
			n_blocks = strtol(argv[++i], NULL, 10);
			continue;
		}
		if (strcmp(argv[i], "--threads") == 0 || strcmp(argv[i], "-t") == 0)
		{
			if (i + 1 == argc) usage();
			n_threads = strtol(argv[++i], NULL, 10);
			continue;
		}
		if (strcmp(argv[i], "--max-pass") == 0 || strcmp(argv[i], "-mp") == 0)
		{
			if (i + 1 == argc) usage();
			max_pass = strtol(argv[++i], NULL, 10);
			continue;
		}
		if (strcmp(argv[i], "--max-decode") == 0 || strcmp(argv[i], "-md") == 0)
		{
			if (i + 1 == argc) usage();
			max_decode = strtol(argv[++i], NULL, 10);
			continue;
		}
		if (strcmp(argv[i], "--suffix") == 0 || strcmp(argv[i], "-s") == 0)
		{
			if (i + 1 == argc) usage();
			suffix_length = strtol(argv[++i], NULL, 10);
			continue;
		}

		//fall through

		if (!coeff_file)
		{
			coeff_file = argv[i];
			continue;
		}
		//anything else
		usage();
	}
	if (!coeff_file)
		usage();
	if (!pass_file)
		usage();

	int max_batch = n_blocks * n_threads;

	char * password_buffer;
	char * result_buffer;
	short * coeff_buffer;
	int * perm_buffer;

	short *coeff = 0;                      // coefficent dump from a JPEG file
	int    coeff_len;
	int    max_msg_len;                    // max legit message length
	hipError_t cudaStatus;

	clock_t start, end;
	float seconds;
	start = clock();

	FILE * fp;
	char line[256];
	size_t len = 0;
	int read;

	printf("setting gpu to %d\n", gpu_id);
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(gpu_id);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU and drivers installed?");
		return 1;
	}

	// load coefs and copy to GPU
	printf("loading coeff file %s\n", coeff_file);
	load_coeff(coeff_file, &coeff, &coeff_len, &max_msg_len);

	printf("allocating memory for batches of %d * (%d coeff + %d pass + %d decode)\n", max_batch, coeff_len, max_pass, max_decode + 1);
	cudaStatus = hipMalloc(&coeff_buffer, (sizeof(short) * coeff_len)); CUDA_ERR_CHECK;
	cudaStatus = hipMemcpy(coeff_buffer, coeff, coeff_len * sizeof(short), hipMemcpyHostToDevice); CUDA_ERR_CHECK;

	// allocate room for passwords and results
	cudaStatus = hipMalloc(&perm_buffer, (sizeof(int) * max_batch * coeff_len)); CUDA_ERR_CHECK;
	cudaStatus = hipMalloc(&password_buffer, (sizeof(char) * max_batch * max_pass)); CUDA_ERR_CHECK;
	cudaStatus = hipMalloc(&result_buffer, (sizeof(char) * max_batch * (max_decode + 1))); CUDA_ERR_CHECK;

	char * passwords = (char *)malloc(max_batch * max_pass * sizeof(char));
	char * results = (char *)malloc(max_batch * max_decode * sizeof(char));

	printf("opening password file %s\n", pass_file);
	fp = fopen(pass_file, "r");
	if (fp == NULL)
	{
		printf("could not open %s", pass_file);
		exit(EXIT_FAILURE);
	}

	int pass_count = 0;
	int processed = 0;

	if (skip > 0) {
		printf("skipping %d lines\n", skip);
	}

	while (skip-- > 0 && getline(line, &len, fp)) { ; }

	while ((read = getline(line, &len, fp)) != -1) {
//		printf("Retrieved line of length %d :\n", read);
//		printf("%s", line);
		int pass_len = read;
		int off = 0;
		while (pass_len >= suffix_length) {
//			printf("%s %d %d\n", &line[off], pass_len, max_pass);
			if (pass_len < max_pass) {
				memset(&(passwords[pass_count * max_pass]), '\0', max_pass);
				strncpy(&(passwords[pass_count * max_pass]), &line[off], pass_len);
				pass_count++;
				if (pass_count >= max_batch) {
					// printf("batch ready %d\n", pass_count);

					// batch is ready for processing
					cudaStatus = hipMemcpy(password_buffer, passwords, max_batch * max_pass, hipMemcpyHostToDevice); CUDA_ERR_CHECK;
					decode << <n_blocks, n_threads >> > (coeff_buffer, coeff_len, perm_buffer, password_buffer, result_buffer, max_msg_len, max_pass, max_decode);
					cudaStatus = hipGetLastError(); CUDA_ERR_CHECK;
					cudaStatus = hipDeviceSynchronize(); CUDA_ERR_CHECK;
					pass_count = 0;

					processed += max_batch;
					end = clock();
					seconds = end - start; // time difference is now a float
					seconds /= CLOCKS_PER_SEC; // this division is now floating point
					printf("processed %d pass in %.02f seconds = %.02f pass/sec @ %s\n", processed, seconds, processed / seconds, line);
				}
			}
//			else { printf("skipping %s,too long\n", &line[off]); }
			if (suffix_length == 0) { 
				// 0 means no suffixing
				pass_len = 0;
			}
			pass_len--;
			off++;
		}
	}

	if (pass_count > 0) {
		processed += pass_count;
		while (pass_count < max_batch) {
			memset(&(passwords[pass_count * max_pass]), '\0', max_pass);
			pass_count++;
		}
		cudaStatus = hipMemcpy(password_buffer, passwords, max_batch * max_pass, hipMemcpyHostToDevice); CUDA_ERR_CHECK;
		decode << <n_blocks, n_threads >> > (coeff_buffer, coeff_len, perm_buffer, password_buffer, result_buffer, max_msg_len, max_pass, max_decode);
		cudaStatus = hipGetLastError(); CUDA_ERR_CHECK;
		cudaStatus = hipDeviceSynchronize(); CUDA_ERR_CHECK;
		end = clock();
		seconds = end - start; // time difference is now a float
		seconds /= CLOCKS_PER_SEC; // this division is now floating point
		printf("processed %d pass in %.02f seconds = %.02f pass/sec complete\n", processed, seconds, processed / seconds);
	}

	printf("done\n");

	fclose(fp);

	hipFree(coeff_buffer);
	hipFree(perm_buffer);
	hipFree(password_buffer);
	hipFree(result_buffer);

	return 1;
}

int load_coeff(char* filename, short** coeffptr, int* coeff_len, int *max_msg_len)
{
	FILE *fp;
	short *coeff = 0;

	fp = fopen(filename, "rb");
	if (!fp)
	{
		fputs("File not found\n", stderr);
		return 1;
	}
	fseek(fp, 0, SEEK_END);
	*coeff_len = ftell(fp) / 2;
	rewind(fp);

	printf("File: %s   %i bytes.\n", filename, *coeff_len * 2);

	coeff = (short *)malloc(*coeff_len * sizeof(short));
	*coeffptr = coeff; //export the pointer

	if (fread(coeff, 2, *coeff_len, fp) != *coeff_len)
	{
		fputs("File error\n", stderr);
		return 1;
	}

	*max_msg_len = 0;
	for (int i = 0; i < *coeff_len; i++)
		if ((i % 64 != 0) & (coeff[i] != 0))
			(*max_msg_len)++;

	printf("Max theoretical message length: %i\n", *max_msg_len);
	return 0;
}

int getline(char line[256], size_t *n, FILE *stream)
{
	char *ptr;
	size_t len;

	if (ferror(stream))
		return -1;

	if (feof(stream))
		return -1;

	fgets(line, 256, stream);

	ptr = strchr(line, '\n');
	if (ptr)
		*ptr = '\0';

	len = strlen(line);

	return((int)len);
}
