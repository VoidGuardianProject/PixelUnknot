/* /////////////// DISCLAIMER/////////////////////////////////
   This software is provided by the author and
   contributors ``as is'' and any express or implied
   warranties, including, but not limited to, the
   implied warranties of merchantability and
   fitness for a particular purpose are dis-
   claimed. In no event shall the author or con-
   tributors be liable for any direct, indirect,
   incidental, special, exemplary, or consequen-
   tial damages (including, but not limited to,
   procurement of substitute goods or services;
   loss of use, data, or profits; or business
   interruption) however caused and on any
   theory of liability, whether in contract,
   strict liability, or tort (including negligence
   or otherwise) arising in any way out of the use
   of this software, even if advised of the poss-
   ibility of such damage.
//////////////////////////////////////////////////////*/

// important notes on this:
// input buf unused bytes needs to be set to zero
// input buf needs to be in algorithm native byte order (md5 = LE, sha1 = BE, etc)
// input buf needs to be 64 byte aligned when using md5_update()
//#include "hash_common.cu"
//#include "hash_functions.cu"


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

typedef uint8_t  u8;
typedef uint16_t u16;
typedef uint32_t   u32;
typedef uint64_t  u64;

#define IS_NV 1
#define CUDA_ARCH 350

/*
__device__ uint32_t rotate(uint32_t input, unsigned amount)
{
	// With constant amount, the left/right masks are constants
	uint32_t rmask = 0xFF >> ((8 - amount) & 7);
	rmask = (rmask << 24 | rmask << 16 | rmask << 8 | rmask);
	uint32_t lmask = ~rmask;

	uint32_t lshift = input << amount;
	lshift &= lmask;
	if (amount == 1) {  // special case left-shift by 1 using an in-lane add instead of shift&mask
		lshift = __vadd4(input, input);
	}
	uint32_t rshift = input >> ((8 - amount) & 7);
	rshift &= rmask;

	uint32_t rotated = lshift | rshift;
	return rotated;
}
*/

__device__ u32 rotate(u32 n, unsigned int c)
{
	const unsigned int mask = (CHAR_BIT * sizeof(n) - 1);  // assumes width is a power of 2.

	// assert ( (c<=mask) &&"rotate by type width or more");
	c &= mask;
	return (n << c) | (n >> ((-c)&mask));
}
/*
__device__ u32 rotate(u32 var, u32 hops) {
	return (var << hops) | (var >> ((32 - hops) & 31));
}
*/
__device__ u32 rotl32_S (const u32 a, const u32 n)
{
  return rotate (a, n);
}

__device__ u32 hc_add3_S(const u32 a, const u32 b, const u32 c)
{
	return a + b + c;
}

#define SHA1_F0(x,y,z)  ((z) ^ ((x) & ((y) ^ (z))))
#define SHA1_F1(x,y,z)  ((x) ^ (y) ^ (z))
#define SHA1_F2(x,y,z)  (((x) & (y)) | ((z) & ((x) ^ (y))))
#define SHA1_F0o(x,y,z) (SHA1_F0 ((x), (y), (z)))
#define SHA1_F2o(x,y,z) (SHA1_F2 ((x), (y), (z)))

#define SHA1_STEP_S(f,a,b,c,d,e,x)    \
{                                     \
  e += K;                             \
  e  = hc_add3_S (e, x, f (b, c, d)); \
  e += rotl32_S (a,  5u);             \
  b  = rotl32_S (b, 30u);             \
}

#define SHA1_STEP(f,a,b,c,d,e,x)    \
{                                   \
  e += K;                           \
  e  = hc_add3 (e, x, f (b, c, d)); \
  e += rotl32 (a,  5u);             \
  b  = rotl32 (b, 30u);             \
}
typedef enum sha1_constants
{
	SHA1M_A = 0x67452301,
	SHA1M_B = 0xefcdab89,
	SHA1M_C = 0x98badcfe,
	SHA1M_D = 0x10325476,
	SHA1M_E = 0xc3d2e1f0,

	SHA1C00 = 0x5a827999,
	SHA1C01 = 0x6ed9eba1,
	SHA1C02 = 0x8f1bbcdc,
	SHA1C03 = 0xca62c1d6u

} sha1_constants_t;

typedef struct sha1_ctx
{
  u32 h[5];

  u32 w0[4];
  u32 w1[4];
  u32 w2[4];
  u32 w3[4];

  int len;

} sha1_ctx_t;

__device__ void sha1_transform (const u32 *w0, const u32 *w1, const u32 *w2, const u32 *w3, u32 *digest)
{
	/*
	printf("hash sha1 transform @ %02x%02x%02x%02x%02x ", digest[0], digest[1], digest[2], digest[3], digest[4]);
	for (int i = 0; i < 16; i++)
	{
		printf("%02x", ((w0[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
	}
	for (int i = 0; i < 16; i++)
	{
		printf("%02x", ((w1[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
	}
	for (int i = 0; i < 16; i++)
	{
		printf("%02x", ((w2[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
	}
	for (int i = 0; i < 16; i++)
	{
		printf("%02x", ((w3[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
	}
	printf("\n");
	*/
  u32 a = digest[0];
  u32 b = digest[1];
  u32 c = digest[2];
  u32 d = digest[3];
  u32 e = digest[4];

  u32 w0_t = w0[0];
  u32 w1_t = w0[1];
  u32 w2_t = w0[2];
  u32 w3_t = w0[3];
  u32 w4_t = w1[0];
  u32 w5_t = w1[1];
  u32 w6_t = w1[2];
  u32 w7_t = w1[3];
  u32 w8_t = w2[0];
  u32 w9_t = w2[1];
  u32 wa_t = w2[2];
  u32 wb_t = w2[3];
  u32 wc_t = w3[0];
  u32 wd_t = w3[1];
  u32 we_t = w3[2];
  u32 wf_t = w3[3];

  #define K SHA1C00

  SHA1_STEP_S (SHA1_F0o, a, b, c, d, e, w0_t);
  SHA1_STEP_S (SHA1_F0o, e, a, b, c, d, w1_t);
  SHA1_STEP_S (SHA1_F0o, d, e, a, b, c, w2_t);
  SHA1_STEP_S (SHA1_F0o, c, d, e, a, b, w3_t);
  SHA1_STEP_S (SHA1_F0o, b, c, d, e, a, w4_t);
  SHA1_STEP_S (SHA1_F0o, a, b, c, d, e, w5_t);
  SHA1_STEP_S (SHA1_F0o, e, a, b, c, d, w6_t);
  SHA1_STEP_S (SHA1_F0o, d, e, a, b, c, w7_t);
  SHA1_STEP_S (SHA1_F0o, c, d, e, a, b, w8_t);
  SHA1_STEP_S (SHA1_F0o, b, c, d, e, a, w9_t);
  SHA1_STEP_S (SHA1_F0o, a, b, c, d, e, wa_t);
  SHA1_STEP_S (SHA1_F0o, e, a, b, c, d, wb_t);
  SHA1_STEP_S (SHA1_F0o, d, e, a, b, c, wc_t);
  SHA1_STEP_S (SHA1_F0o, c, d, e, a, b, wd_t);
  SHA1_STEP_S (SHA1_F0o, b, c, d, e, a, we_t);
  SHA1_STEP_S (SHA1_F0o, a, b, c, d, e, wf_t);
  w0_t = rotl32_S ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP_S (SHA1_F0o, e, a, b, c, d, w0_t);
  w1_t = rotl32_S ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP_S (SHA1_F0o, d, e, a, b, c, w1_t);
  w2_t = rotl32_S ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP_S (SHA1_F0o, c, d, e, a, b, w2_t);
  w3_t = rotl32_S ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP_S (SHA1_F0o, b, c, d, e, a, w3_t);

  #undef K
  #define K SHA1C01

  w4_t = rotl32_S ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, w4_t);
  w5_t = rotl32_S ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, w5_t);
  w6_t = rotl32_S ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, w6_t);
  w7_t = rotl32_S ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, w7_t);
  w8_t = rotl32_S ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, w8_t);
  w9_t = rotl32_S ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, w9_t);
  wa_t = rotl32_S ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, wa_t);
  wb_t = rotl32_S ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, wb_t);
  wc_t = rotl32_S ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, wc_t);
  wd_t = rotl32_S ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, wd_t);
  we_t = rotl32_S ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, we_t);
  wf_t = rotl32_S ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, wf_t);
  w0_t = rotl32_S ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, w0_t);
  w1_t = rotl32_S ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, w1_t);
  w2_t = rotl32_S ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, w2_t);
  w3_t = rotl32_S ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, w3_t);
  w4_t = rotl32_S ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, w4_t);
  w5_t = rotl32_S ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, w5_t);
  w6_t = rotl32_S ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, w6_t);
  w7_t = rotl32_S ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, w7_t);

  #undef K
  #define K SHA1C02

  w8_t = rotl32_S ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP_S (SHA1_F2o, a, b, c, d, e, w8_t);
  w9_t = rotl32_S ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP_S (SHA1_F2o, e, a, b, c, d, w9_t);
  wa_t = rotl32_S ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP_S (SHA1_F2o, d, e, a, b, c, wa_t);
  wb_t = rotl32_S ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP_S (SHA1_F2o, c, d, e, a, b, wb_t);
  wc_t = rotl32_S ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP_S (SHA1_F2o, b, c, d, e, a, wc_t);
  wd_t = rotl32_S ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP_S (SHA1_F2o, a, b, c, d, e, wd_t);
  we_t = rotl32_S ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP_S (SHA1_F2o, e, a, b, c, d, we_t);
  wf_t = rotl32_S ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP_S (SHA1_F2o, d, e, a, b, c, wf_t);
  w0_t = rotl32_S ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP_S (SHA1_F2o, c, d, e, a, b, w0_t);
  w1_t = rotl32_S ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP_S (SHA1_F2o, b, c, d, e, a, w1_t);
  w2_t = rotl32_S ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP_S (SHA1_F2o, a, b, c, d, e, w2_t);
  w3_t = rotl32_S ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP_S (SHA1_F2o, e, a, b, c, d, w3_t);
  w4_t = rotl32_S ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP_S (SHA1_F2o, d, e, a, b, c, w4_t);
  w5_t = rotl32_S ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP_S (SHA1_F2o, c, d, e, a, b, w5_t);
  w6_t = rotl32_S ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP_S (SHA1_F2o, b, c, d, e, a, w6_t);
  w7_t = rotl32_S ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP_S (SHA1_F2o, a, b, c, d, e, w7_t);
  w8_t = rotl32_S ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP_S (SHA1_F2o, e, a, b, c, d, w8_t);
  w9_t = rotl32_S ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP_S (SHA1_F2o, d, e, a, b, c, w9_t);
  wa_t = rotl32_S ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP_S (SHA1_F2o, c, d, e, a, b, wa_t);
  wb_t = rotl32_S ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP_S (SHA1_F2o, b, c, d, e, a, wb_t);

  #undef K
  #define K SHA1C03

  wc_t = rotl32_S ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, wc_t);
  wd_t = rotl32_S ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, wd_t);
  we_t = rotl32_S ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, we_t);
  wf_t = rotl32_S ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, wf_t);
  w0_t = rotl32_S ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, w0_t);
  w1_t = rotl32_S ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, w1_t);
  w2_t = rotl32_S ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, w2_t);
  w3_t = rotl32_S ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, w3_t);
  w4_t = rotl32_S ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, w4_t);
  w5_t = rotl32_S ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, w5_t);
  w6_t = rotl32_S ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, w6_t);
  w7_t = rotl32_S ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, w7_t);
  w8_t = rotl32_S ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, w8_t);
  w9_t = rotl32_S ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, w9_t);
  wa_t = rotl32_S ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, wa_t);
  wb_t = rotl32_S ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP_S (SHA1_F1, a, b, c, d, e, wb_t);
  wc_t = rotl32_S ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP_S (SHA1_F1, e, a, b, c, d, wc_t);
  wd_t = rotl32_S ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP_S (SHA1_F1, d, e, a, b, c, wd_t);
  we_t = rotl32_S ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP_S (SHA1_F1, c, d, e, a, b, we_t);
  wf_t = rotl32_S ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP_S (SHA1_F1, b, c, d, e, a, wf_t);

  #undef K

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
  digest[4] += e;
}

__device__ void sha1_init (sha1_ctx_t *ctx)
{
  ctx->h[0] = SHA1M_A;
  ctx->h[1] = SHA1M_B;
  ctx->h[2] = SHA1M_C;
  ctx->h[3] = SHA1M_D;
  ctx->h[4] = SHA1M_E;

  ctx->w0[0] = 0;
  ctx->w0[1] = 0;
  ctx->w0[2] = 0;
  ctx->w0[3] = 0;
  ctx->w1[0] = 0;
  ctx->w1[1] = 0;
  ctx->w1[2] = 0;
  ctx->w1[3] = 0;
  ctx->w2[0] = 0;
  ctx->w2[1] = 0;
  ctx->w2[2] = 0;
  ctx->w2[3] = 0;
  ctx->w3[0] = 0;
  ctx->w3[1] = 0;
  ctx->w3[2] = 0;
  ctx->w3[3] = 0;

  ctx->len = 0;
}

__device__ u32 hc_byte_perm_S(const u32 a, const u32 b, const u32 c)
{
	u32 r;

	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));

	return r;
}

__device__ u32 hc_bytealign_S (const u32 a, const u32 b, const u32 c)
{
  u32 r;

  #if CUDA_ARCH >= 350

  asm ("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(r) : "r"(b), "r"(a), "r"((c & 3) * 8));

  #else

  r = hc_byte_perm_S (b, a, (0x76543210 >> ((c & 3) * 4)) & 0xffff);

  #endif

  return r;
}

__device__ void switch_buffer_by_offset_be_S (u32 *w0, u32 *w1, u32 *w2, u32 *w3, const u32 offset)
{
  const int offset_switch = offset / 4;

  #if (defined IS_AMD && AMD_GCN < 3) || defined IS_GENERIC
  switch (offset_switch)
  {
    case  0:
      w3[3] = hc_bytealign_S (w3[2], w3[3], offset);
      w3[2] = hc_bytealign_S (w3[1], w3[2], offset);
      w3[1] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[0] = hc_bytealign_S (w2[3], w3[0], offset);
      w2[3] = hc_bytealign_S (w2[2], w2[3], offset);
      w2[2] = hc_bytealign_S (w2[1], w2[2], offset);
      w2[1] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[0] = hc_bytealign_S (w1[3], w2[0], offset);
      w1[3] = hc_bytealign_S (w1[2], w1[3], offset);
      w1[2] = hc_bytealign_S (w1[1], w1[2], offset);
      w1[1] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w0[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w0[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w0[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[0] = hc_bytealign_S (    0, w0[0], offset);

      break;

    case  1:
      w3[3] = hc_bytealign_S (w3[1], w3[2], offset);
      w3[2] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[1] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[0] = hc_bytealign_S (w2[2], w2[3], offset);
      w2[3] = hc_bytealign_S (w2[1], w2[2], offset);
      w2[2] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[1] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[0] = hc_bytealign_S (w1[2], w1[3], offset);
      w1[3] = hc_bytealign_S (w1[1], w1[2], offset);
      w1[2] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[1] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w0[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w0[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[1] = hc_bytealign_S (    0, w0[0], offset);
      w0[0] = 0;

      break;

    case  2:
      w3[3] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[2] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[1] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[0] = hc_bytealign_S (w2[1], w2[2], offset);
      w2[3] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[2] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[1] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[0] = hc_bytealign_S (w1[1], w1[2], offset);
      w1[3] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[2] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[1] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w0[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[2] = hc_bytealign_S (    0, w0[0], offset);
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  3:
      w3[3] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[2] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[1] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[0] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[3] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[2] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[1] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[0] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[3] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[2] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[1] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[3] = hc_bytealign_S (    0, w0[0], offset);
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  4:
      w3[3] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[2] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[1] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[0] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[3] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[2] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[1] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[0] = hc_bytealign_S (    0, w0[0], offset);
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  5:
      w3[3] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[2] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[1] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[0] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[3] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[2] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[1] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[1] = hc_bytealign_S (    0, w0[0], offset);
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  6:
      w3[3] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[2] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[1] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[0] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[3] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[2] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[1] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[2] = hc_bytealign_S (    0, w0[0], offset);
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  7:
      w3[3] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[2] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[1] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[0] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[3] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[2] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[1] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[3] = hc_bytealign_S (    0, w0[0], offset);
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  8:
      w3[3] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[2] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[1] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[0] = hc_bytealign_S (    0, w0[0], offset);
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  9:
      w3[3] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[2] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[1] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[1] = hc_bytealign_S (    0, w0[0], offset);
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 10:
      w3[3] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[2] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[1] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[2] = hc_bytealign_S (    0, w0[0], offset);
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 11:
      w3[3] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[2] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[1] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[3] = hc_bytealign_S (    0, w0[0], offset);
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 12:
      w3[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[0] = hc_bytealign_S (    0, w0[0], offset);
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 13:
      w3[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[1] = hc_bytealign_S (    0, w0[0], offset);
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 14:
      w3[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[2] = hc_bytealign_S (    0, w0[0], offset);
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 15:
      w3[3] = hc_bytealign_S (    0, w0[0], offset);
      w3[2] = 0;
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;
  }
  #endif

  #if (defined IS_AMD && AMD_GCN >= 3) || defined IS_NV

  #if defined IS_NV
  const int selector = (0x76543210 >> ((offset & 3) * 4)) & 0xffff;
  #endif

  #if defined IS_AMD
  const int selector = 0x0706050403020100 >> ((offset & 3) * 8);
  #endif

  switch (offset_switch)
  {
    case  0:
      w3[3] = hc_byte_perm_S (w3[3], w3[2], selector);
      w3[2] = hc_byte_perm_S (w3[2], w3[1], selector);
      w3[1] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[0] = hc_byte_perm_S (w3[0], w2[3], selector);
      w2[3] = hc_byte_perm_S (w2[3], w2[2], selector);
      w2[2] = hc_byte_perm_S (w2[2], w2[1], selector);
      w2[1] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[0] = hc_byte_perm_S (w2[0], w1[3], selector);
      w1[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      w1[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      w1[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w0[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w0[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w0[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[0] = hc_byte_perm_S (w0[0],     0, selector);

      break;

    case  1:
      w3[3] = hc_byte_perm_S (w3[2], w3[1], selector);
      w3[2] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[1] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[0] = hc_byte_perm_S (w2[3], w2[2], selector);
      w2[3] = hc_byte_perm_S (w2[2], w2[1], selector);
      w2[2] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[1] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[0] = hc_byte_perm_S (w1[3], w1[2], selector);
      w1[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      w1[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w0[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w0[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[1] = hc_byte_perm_S (w0[0],     0, selector);
      w0[0] = 0;

      break;

    case  2:
      w3[3] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[2] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[1] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[0] = hc_byte_perm_S (w2[2], w2[1], selector);
      w2[3] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[2] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[1] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[0] = hc_byte_perm_S (w1[2], w1[1], selector);
      w1[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w0[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[2] = hc_byte_perm_S (w0[0],     0, selector);
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  3:
      w3[3] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[2] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[1] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[0] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[3] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[2] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[1] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[0] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[3] = hc_byte_perm_S (w0[0],     0, selector);
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  4:
      w3[3] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[2] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[1] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[0] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[0] = hc_byte_perm_S (w0[0],     0, selector);
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  5:
      w3[3] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[2] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[1] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[0] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[1] = hc_byte_perm_S (w0[0],     0, selector);
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  6:
      w3[3] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[2] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[1] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[0] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[2] = hc_byte_perm_S (w0[0],     0, selector);
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  7:
      w3[3] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[2] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[1] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[0] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[3] = hc_byte_perm_S (w0[0],     0, selector);
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  8:
      w3[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[0] = hc_byte_perm_S (w0[0],     0, selector);
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  9:
      w3[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[1] = hc_byte_perm_S (w0[0],     0, selector);
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 10:
      w3[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[2] = hc_byte_perm_S (w0[0],     0, selector);
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 11:
      w3[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[3] = hc_byte_perm_S (w0[0],     0, selector);
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 12:
      w3[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[0] = hc_byte_perm_S (w0[0],     0, selector);
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 13:
      w3[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[1] = hc_byte_perm_S (w0[0],     0, selector);
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 14:
      w3[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[2] = hc_byte_perm_S (w0[0],     0, selector);
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 15:
      w3[3] = hc_byte_perm_S (w0[0],     0, selector);
      w3[2] = 0;
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;
  }
  #endif
}

__device__ void switch_buffer_by_offset_carry_be_S (u32 *w0, u32 *w1, u32 *w2, u32 *w3, u32 *c0, u32 *c1, u32 *c2, u32 *c3, const u32 offset)
{
  const int offset_switch = offset / 4;

  #if (defined IS_AMD && AMD_GCN < 3) || defined IS_GENERIC
  switch (offset_switch)
  {
    case  0:
      c0[0] = hc_bytealign_S (w3[3],     0, offset);
      w3[3] = hc_bytealign_S (w3[2], w3[3], offset);
      w3[2] = hc_bytealign_S (w3[1], w3[2], offset);
      w3[1] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[0] = hc_bytealign_S (w2[3], w3[0], offset);
      w2[3] = hc_bytealign_S (w2[2], w2[3], offset);
      w2[2] = hc_bytealign_S (w2[1], w2[2], offset);
      w2[1] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[0] = hc_bytealign_S (w1[3], w2[0], offset);
      w1[3] = hc_bytealign_S (w1[2], w1[3], offset);
      w1[2] = hc_bytealign_S (w1[1], w1[2], offset);
      w1[1] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w0[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w0[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w0[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[0] = hc_bytealign_S (    0, w0[0], offset);

      break;

    case  1:
      c0[1] = hc_bytealign_S (w3[3],     0, offset);
      c0[0] = hc_bytealign_S (w3[2], w3[3], offset);
      w3[3] = hc_bytealign_S (w3[1], w3[2], offset);
      w3[2] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[1] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[0] = hc_bytealign_S (w2[2], w2[3], offset);
      w2[3] = hc_bytealign_S (w2[1], w2[2], offset);
      w2[2] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[1] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[0] = hc_bytealign_S (w1[2], w1[3], offset);
      w1[3] = hc_bytealign_S (w1[1], w1[2], offset);
      w1[2] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[1] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w0[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w0[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[1] = hc_bytealign_S (    0, w0[0], offset);
      w0[0] = 0;

      break;

    case  2:
      c0[2] = hc_bytealign_S (w3[3],     0, offset);
      c0[1] = hc_bytealign_S (w3[2], w3[3], offset);
      c0[0] = hc_bytealign_S (w3[1], w3[2], offset);
      w3[3] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[2] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[1] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[0] = hc_bytealign_S (w2[1], w2[2], offset);
      w2[3] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[2] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[1] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[0] = hc_bytealign_S (w1[1], w1[2], offset);
      w1[3] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[2] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[1] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w0[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[2] = hc_bytealign_S (    0, w0[0], offset);
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  3:
      c0[3] = hc_bytealign_S (w3[3],     0, offset);
      c0[2] = hc_bytealign_S (w3[2], w3[3], offset);
      c0[1] = hc_bytealign_S (w3[1], w3[2], offset);
      c0[0] = hc_bytealign_S (w3[0], w3[1], offset);
      w3[3] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[2] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[1] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[0] = hc_bytealign_S (w2[0], w2[1], offset);
      w2[3] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[2] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[1] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[0] = hc_bytealign_S (w1[0], w1[1], offset);
      w1[3] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[2] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[1] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w0[3] = hc_bytealign_S (    0, w0[0], offset);
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  4:
      c1[0] = hc_bytealign_S (w3[3],     0, offset);
      c0[3] = hc_bytealign_S (w3[2], w3[3], offset);
      c0[2] = hc_bytealign_S (w3[1], w3[2], offset);
      c0[1] = hc_bytealign_S (w3[0], w3[1], offset);
      c0[0] = hc_bytealign_S (w2[3], w3[0], offset);
      w3[3] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[2] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[1] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[0] = hc_bytealign_S (w1[3], w2[0], offset);
      w2[3] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[2] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[1] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w1[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[0] = hc_bytealign_S (    0, w0[0], offset);
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  5:
      c1[1] = hc_bytealign_S (w3[3],     0, offset);
      c1[0] = hc_bytealign_S (w3[2], w3[3], offset);
      c0[3] = hc_bytealign_S (w3[1], w3[2], offset);
      c0[2] = hc_bytealign_S (w3[0], w3[1], offset);
      c0[1] = hc_bytealign_S (w2[3], w3[0], offset);
      c0[0] = hc_bytealign_S (w2[2], w2[3], offset);
      w3[3] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[2] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[1] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[0] = hc_bytealign_S (w1[2], w1[3], offset);
      w2[3] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[2] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[1] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w1[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[1] = hc_bytealign_S (    0, w0[0], offset);
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  6:
      c1[2] = hc_bytealign_S (w3[3],     0, offset);
      c1[1] = hc_bytealign_S (w3[2], w3[3], offset);
      c1[0] = hc_bytealign_S (w3[1], w3[2], offset);
      c0[3] = hc_bytealign_S (w3[0], w3[1], offset);
      c0[2] = hc_bytealign_S (w2[3], w3[0], offset);
      c0[1] = hc_bytealign_S (w2[2], w2[3], offset);
      c0[0] = hc_bytealign_S (w2[1], w2[2], offset);
      w3[3] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[2] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[1] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[0] = hc_bytealign_S (w1[1], w1[2], offset);
      w2[3] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[2] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[1] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w1[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[2] = hc_bytealign_S (    0, w0[0], offset);
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  7:
      c1[3] = hc_bytealign_S (w3[3],     0, offset);
      c1[2] = hc_bytealign_S (w3[2], w3[3], offset);
      c1[1] = hc_bytealign_S (w3[1], w3[2], offset);
      c1[0] = hc_bytealign_S (w3[0], w3[1], offset);
      c0[3] = hc_bytealign_S (w2[3], w3[0], offset);
      c0[2] = hc_bytealign_S (w2[2], w2[3], offset);
      c0[1] = hc_bytealign_S (w2[1], w2[2], offset);
      c0[0] = hc_bytealign_S (w2[0], w2[1], offset);
      w3[3] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[2] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[1] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[0] = hc_bytealign_S (w1[0], w1[1], offset);
      w2[3] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[2] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[1] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w1[3] = hc_bytealign_S (    0, w0[0], offset);
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  8:
      c2[0] = hc_bytealign_S (w3[3],     0, offset);
      c1[3] = hc_bytealign_S (w3[2], w3[3], offset);
      c1[2] = hc_bytealign_S (w3[1], w3[2], offset);
      c1[1] = hc_bytealign_S (w3[0], w3[1], offset);
      c1[0] = hc_bytealign_S (w2[3], w3[0], offset);
      c0[3] = hc_bytealign_S (w2[2], w2[3], offset);
      c0[2] = hc_bytealign_S (w2[1], w2[2], offset);
      c0[1] = hc_bytealign_S (w2[0], w2[1], offset);
      c0[0] = hc_bytealign_S (w1[3], w2[0], offset);
      w3[3] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[2] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[1] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w2[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[0] = hc_bytealign_S (    0, w0[0], offset);
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  9:
      c2[1] = hc_bytealign_S (w3[3],     0, offset);
      c2[0] = hc_bytealign_S (w3[2], w3[3], offset);
      c1[3] = hc_bytealign_S (w3[1], w3[2], offset);
      c1[2] = hc_bytealign_S (w3[0], w3[1], offset);
      c1[1] = hc_bytealign_S (w2[3], w3[0], offset);
      c1[0] = hc_bytealign_S (w2[2], w2[3], offset);
      c0[3] = hc_bytealign_S (w2[1], w2[2], offset);
      c0[2] = hc_bytealign_S (w2[0], w2[1], offset);
      c0[1] = hc_bytealign_S (w1[3], w2[0], offset);
      c0[0] = hc_bytealign_S (w1[2], w1[3], offset);
      w3[3] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[2] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[1] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w2[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[1] = hc_bytealign_S (    0, w0[0], offset);
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 10:
      c2[2] = hc_bytealign_S (w3[3],     0, offset);
      c2[1] = hc_bytealign_S (w3[2], w3[3], offset);
      c2[0] = hc_bytealign_S (w3[1], w3[2], offset);
      c1[3] = hc_bytealign_S (w3[0], w3[1], offset);
      c1[2] = hc_bytealign_S (w2[3], w3[0], offset);
      c1[1] = hc_bytealign_S (w2[2], w2[3], offset);
      c1[0] = hc_bytealign_S (w2[1], w2[2], offset);
      c0[3] = hc_bytealign_S (w2[0], w2[1], offset);
      c0[2] = hc_bytealign_S (w1[3], w2[0], offset);
      c0[1] = hc_bytealign_S (w1[2], w1[3], offset);
      c0[0] = hc_bytealign_S (w1[1], w1[2], offset);
      w3[3] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[2] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[1] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w2[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[2] = hc_bytealign_S (    0, w0[0], offset);
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 11:
      c2[3] = hc_bytealign_S (w3[3],     0, offset);
      c2[2] = hc_bytealign_S (w3[2], w3[3], offset);
      c2[1] = hc_bytealign_S (w3[1], w3[2], offset);
      c2[0] = hc_bytealign_S (w3[0], w3[1], offset);
      c1[3] = hc_bytealign_S (w2[3], w3[0], offset);
      c1[2] = hc_bytealign_S (w2[2], w2[3], offset);
      c1[1] = hc_bytealign_S (w2[1], w2[2], offset);
      c1[0] = hc_bytealign_S (w2[0], w2[1], offset);
      c0[3] = hc_bytealign_S (w1[3], w2[0], offset);
      c0[2] = hc_bytealign_S (w1[2], w1[3], offset);
      c0[1] = hc_bytealign_S (w1[1], w1[2], offset);
      c0[0] = hc_bytealign_S (w1[0], w1[1], offset);
      w3[3] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[2] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[1] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w2[3] = hc_bytealign_S (    0, w0[0], offset);
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 12:
      c3[0] = hc_bytealign_S (w3[3],     0, offset);
      c2[3] = hc_bytealign_S (w3[2], w3[3], offset);
      c2[2] = hc_bytealign_S (w3[1], w3[2], offset);
      c2[1] = hc_bytealign_S (w3[0], w3[1], offset);
      c2[0] = hc_bytealign_S (w2[3], w3[0], offset);
      c1[3] = hc_bytealign_S (w2[2], w2[3], offset);
      c1[2] = hc_bytealign_S (w2[1], w2[2], offset);
      c1[1] = hc_bytealign_S (w2[0], w2[1], offset);
      c1[0] = hc_bytealign_S (w1[3], w2[0], offset);
      c0[3] = hc_bytealign_S (w1[2], w1[3], offset);
      c0[2] = hc_bytealign_S (w1[1], w1[2], offset);
      c0[1] = hc_bytealign_S (w1[0], w1[1], offset);
      c0[0] = hc_bytealign_S (w0[3], w1[0], offset);
      w3[3] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[2] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[1] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[0] = hc_bytealign_S (    0, w0[0], offset);
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 13:
      c3[1] = hc_bytealign_S (w3[3],     0, offset);
      c3[0] = hc_bytealign_S (w3[2], w3[3], offset);
      c2[3] = hc_bytealign_S (w3[1], w3[2], offset);
      c2[2] = hc_bytealign_S (w3[0], w3[1], offset);
      c2[1] = hc_bytealign_S (w2[3], w3[0], offset);
      c2[0] = hc_bytealign_S (w2[2], w2[3], offset);
      c1[3] = hc_bytealign_S (w2[1], w2[2], offset);
      c1[2] = hc_bytealign_S (w2[0], w2[1], offset);
      c1[1] = hc_bytealign_S (w1[3], w2[0], offset);
      c1[0] = hc_bytealign_S (w1[2], w1[3], offset);
      c0[3] = hc_bytealign_S (w1[1], w1[2], offset);
      c0[2] = hc_bytealign_S (w1[0], w1[1], offset);
      c0[1] = hc_bytealign_S (w0[3], w1[0], offset);
      c0[0] = hc_bytealign_S (w0[2], w0[3], offset);
      w3[3] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[2] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[1] = hc_bytealign_S (    0, w0[0], offset);
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 14:
      c3[2] = hc_bytealign_S (w3[3],     0, offset);
      c3[1] = hc_bytealign_S (w3[2], w3[3], offset);
      c3[0] = hc_bytealign_S (w3[1], w3[2], offset);
      c2[3] = hc_bytealign_S (w3[0], w3[1], offset);
      c2[2] = hc_bytealign_S (w2[3], w3[0], offset);
      c2[1] = hc_bytealign_S (w2[2], w2[3], offset);
      c2[0] = hc_bytealign_S (w2[1], w2[2], offset);
      c1[3] = hc_bytealign_S (w2[0], w2[1], offset);
      c1[2] = hc_bytealign_S (w1[3], w2[0], offset);
      c1[1] = hc_bytealign_S (w1[2], w1[3], offset);
      c1[0] = hc_bytealign_S (w1[1], w1[2], offset);
      c0[3] = hc_bytealign_S (w1[0], w1[1], offset);
      c0[2] = hc_bytealign_S (w0[3], w1[0], offset);
      c0[1] = hc_bytealign_S (w0[2], w0[3], offset);
      c0[0] = hc_bytealign_S (w0[1], w0[2], offset);
      w3[3] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[2] = hc_bytealign_S (    0, w0[0], offset);
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 15:
      c3[3] = hc_bytealign_S (w3[3],     0, offset);
      c3[2] = hc_bytealign_S (w3[2], w3[3], offset);
      c3[1] = hc_bytealign_S (w3[1], w3[2], offset);
      c3[0] = hc_bytealign_S (w3[0], w3[1], offset);
      c2[3] = hc_bytealign_S (w2[3], w3[0], offset);
      c2[2] = hc_bytealign_S (w2[2], w2[3], offset);
      c2[1] = hc_bytealign_S (w2[1], w2[2], offset);
      c2[0] = hc_bytealign_S (w2[0], w2[1], offset);
      c1[3] = hc_bytealign_S (w1[3], w2[0], offset);
      c1[2] = hc_bytealign_S (w1[2], w1[3], offset);
      c1[1] = hc_bytealign_S (w1[1], w1[2], offset);
      c1[0] = hc_bytealign_S (w1[0], w1[1], offset);
      c0[3] = hc_bytealign_S (w0[3], w1[0], offset);
      c0[2] = hc_bytealign_S (w0[2], w0[3], offset);
      c0[1] = hc_bytealign_S (w0[1], w0[2], offset);
      c0[0] = hc_bytealign_S (w0[0], w0[1], offset);
      w3[3] = hc_bytealign_S (    0, w0[0], offset);
      w3[2] = 0;
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;
  }
  #endif

  #if (defined IS_AMD && AMD_GCN >= 3) || defined IS_NV

  #if defined IS_NV
  const int selector = (0x76543210 >> ((offset & 3) * 4)) & 0xffff;
  #endif

  #if defined IS_AMD
  const int selector = 0x0706050403020100 >> ((offset & 3) * 8);
  #endif

  switch (offset_switch)
  {
    case  0:
      c0[0] = hc_byte_perm_S (    0, w3[3], selector);
      w3[3] = hc_byte_perm_S (w3[3], w3[2], selector);
      w3[2] = hc_byte_perm_S (w3[2], w3[1], selector);
      w3[1] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[0] = hc_byte_perm_S (w3[0], w2[3], selector);
      w2[3] = hc_byte_perm_S (w2[3], w2[2], selector);
      w2[2] = hc_byte_perm_S (w2[2], w2[1], selector);
      w2[1] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[0] = hc_byte_perm_S (w2[0], w1[3], selector);
      w1[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      w1[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      w1[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w0[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w0[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w0[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[0] = hc_byte_perm_S (w0[0],     0, selector);

      break;

    case  1:
      c0[1] = hc_byte_perm_S (    0, w3[3], selector);
      c0[0] = hc_byte_perm_S (w3[3], w3[2], selector);
      w3[3] = hc_byte_perm_S (w3[2], w3[1], selector);
      w3[2] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[1] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[0] = hc_byte_perm_S (w2[3], w2[2], selector);
      w2[3] = hc_byte_perm_S (w2[2], w2[1], selector);
      w2[2] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[1] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[0] = hc_byte_perm_S (w1[3], w1[2], selector);
      w1[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      w1[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w0[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w0[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[1] = hc_byte_perm_S (w0[0],     0, selector);
      w0[0] = 0;

      break;

    case  2:
      c0[2] = hc_byte_perm_S (    0, w3[3], selector);
      c0[1] = hc_byte_perm_S (w3[3], w3[2], selector);
      c0[0] = hc_byte_perm_S (w3[2], w3[1], selector);
      w3[3] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[2] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[1] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[0] = hc_byte_perm_S (w2[2], w2[1], selector);
      w2[3] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[2] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[1] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[0] = hc_byte_perm_S (w1[2], w1[1], selector);
      w1[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w0[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[2] = hc_byte_perm_S (w0[0],     0, selector);
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  3:
      c0[3] = hc_byte_perm_S (    0, w3[3], selector);
      c0[2] = hc_byte_perm_S (w3[3], w3[2], selector);
      c0[1] = hc_byte_perm_S (w3[2], w3[1], selector);
      c0[0] = hc_byte_perm_S (w3[1], w3[0], selector);
      w3[3] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[2] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[1] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[0] = hc_byte_perm_S (w2[1], w2[0], selector);
      w2[3] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[2] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[1] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[0] = hc_byte_perm_S (w1[1], w1[0], selector);
      w1[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w0[3] = hc_byte_perm_S (w0[0],     0, selector);
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  4:
      c1[0] = hc_byte_perm_S (    0, w3[3], selector);
      c0[3] = hc_byte_perm_S (w3[3], w3[2], selector);
      c0[2] = hc_byte_perm_S (w3[2], w3[1], selector);
      c0[1] = hc_byte_perm_S (w3[1], w3[0], selector);
      c0[0] = hc_byte_perm_S (w3[0], w2[3], selector);
      w3[3] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[2] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[1] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[0] = hc_byte_perm_S (w2[0], w1[3], selector);
      w2[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w1[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[0] = hc_byte_perm_S (w0[0],     0, selector);
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  5:
      c1[1] = hc_byte_perm_S (    0, w3[3], selector);
      c1[0] = hc_byte_perm_S (w3[3], w3[2], selector);
      c0[3] = hc_byte_perm_S (w3[2], w3[1], selector);
      c0[2] = hc_byte_perm_S (w3[1], w3[0], selector);
      c0[1] = hc_byte_perm_S (w3[0], w2[3], selector);
      c0[0] = hc_byte_perm_S (w2[3], w2[2], selector);
      w3[3] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[2] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[1] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[0] = hc_byte_perm_S (w1[3], w1[2], selector);
      w2[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w1[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[1] = hc_byte_perm_S (w0[0],     0, selector);
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  6:
      c1[2] = hc_byte_perm_S (    0, w3[3], selector);
      c1[1] = hc_byte_perm_S (w3[3], w3[2], selector);
      c1[0] = hc_byte_perm_S (w3[2], w3[1], selector);
      c0[3] = hc_byte_perm_S (w3[1], w3[0], selector);
      c0[2] = hc_byte_perm_S (w3[0], w2[3], selector);
      c0[1] = hc_byte_perm_S (w2[3], w2[2], selector);
      c0[0] = hc_byte_perm_S (w2[2], w2[1], selector);
      w3[3] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[2] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[1] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[0] = hc_byte_perm_S (w1[2], w1[1], selector);
      w2[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w1[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[2] = hc_byte_perm_S (w0[0],     0, selector);
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  7:
      c1[3] = hc_byte_perm_S (    0, w3[3], selector);
      c1[2] = hc_byte_perm_S (w3[3], w3[2], selector);
      c1[1] = hc_byte_perm_S (w3[2], w3[1], selector);
      c1[0] = hc_byte_perm_S (w3[1], w3[0], selector);
      c0[3] = hc_byte_perm_S (w3[0], w2[3], selector);
      c0[2] = hc_byte_perm_S (w2[3], w2[2], selector);
      c0[1] = hc_byte_perm_S (w2[2], w2[1], selector);
      c0[0] = hc_byte_perm_S (w2[1], w2[0], selector);
      w3[3] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[2] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[1] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[0] = hc_byte_perm_S (w1[1], w1[0], selector);
      w2[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w1[3] = hc_byte_perm_S (w0[0],     0, selector);
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  8:
      c2[0] = hc_byte_perm_S (    0, w3[3], selector);
      c1[3] = hc_byte_perm_S (w3[3], w3[2], selector);
      c1[2] = hc_byte_perm_S (w3[2], w3[1], selector);
      c1[1] = hc_byte_perm_S (w3[1], w3[0], selector);
      c1[0] = hc_byte_perm_S (w3[0], w2[3], selector);
      c0[3] = hc_byte_perm_S (w2[3], w2[2], selector);
      c0[2] = hc_byte_perm_S (w2[2], w2[1], selector);
      c0[1] = hc_byte_perm_S (w2[1], w2[0], selector);
      c0[0] = hc_byte_perm_S (w2[0], w1[3], selector);
      w3[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w2[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[0] = hc_byte_perm_S (w0[0],     0, selector);
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case  9:
      c2[1] = hc_byte_perm_S (    0, w3[3], selector);
      c2[0] = hc_byte_perm_S (w3[3], w3[2], selector);
      c1[3] = hc_byte_perm_S (w3[2], w3[1], selector);
      c1[2] = hc_byte_perm_S (w3[1], w3[0], selector);
      c1[1] = hc_byte_perm_S (w3[0], w2[3], selector);
      c1[0] = hc_byte_perm_S (w2[3], w2[2], selector);
      c0[3] = hc_byte_perm_S (w2[2], w2[1], selector);
      c0[2] = hc_byte_perm_S (w2[1], w2[0], selector);
      c0[1] = hc_byte_perm_S (w2[0], w1[3], selector);
      c0[0] = hc_byte_perm_S (w1[3], w1[2], selector);
      w3[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w2[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[1] = hc_byte_perm_S (w0[0],     0, selector);
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 10:
      c2[2] = hc_byte_perm_S (    0, w3[3], selector);
      c2[1] = hc_byte_perm_S (w3[3], w3[2], selector);
      c2[0] = hc_byte_perm_S (w3[2], w3[1], selector);
      c1[3] = hc_byte_perm_S (w3[1], w3[0], selector);
      c1[2] = hc_byte_perm_S (w3[0], w2[3], selector);
      c1[1] = hc_byte_perm_S (w2[3], w2[2], selector);
      c1[0] = hc_byte_perm_S (w2[2], w2[1], selector);
      c0[3] = hc_byte_perm_S (w2[1], w2[0], selector);
      c0[2] = hc_byte_perm_S (w2[0], w1[3], selector);
      c0[1] = hc_byte_perm_S (w1[3], w1[2], selector);
      c0[0] = hc_byte_perm_S (w1[2], w1[1], selector);
      w3[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w2[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[2] = hc_byte_perm_S (w0[0],     0, selector);
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 11:
      c2[3] = hc_byte_perm_S (    0, w3[3], selector);
      c2[2] = hc_byte_perm_S (w3[3], w3[2], selector);
      c2[1] = hc_byte_perm_S (w3[2], w3[1], selector);
      c2[0] = hc_byte_perm_S (w3[1], w3[0], selector);
      c1[3] = hc_byte_perm_S (w3[0], w2[3], selector);
      c1[2] = hc_byte_perm_S (w2[3], w2[2], selector);
      c1[1] = hc_byte_perm_S (w2[2], w2[1], selector);
      c1[0] = hc_byte_perm_S (w2[1], w2[0], selector);
      c0[3] = hc_byte_perm_S (w2[0], w1[3], selector);
      c0[2] = hc_byte_perm_S (w1[3], w1[2], selector);
      c0[1] = hc_byte_perm_S (w1[2], w1[1], selector);
      c0[0] = hc_byte_perm_S (w1[1], w1[0], selector);
      w3[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w2[3] = hc_byte_perm_S (w0[0],     0, selector);
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 12:
      c3[0] = hc_byte_perm_S (    0, w3[3], selector);
      c2[3] = hc_byte_perm_S (w3[3], w3[2], selector);
      c2[2] = hc_byte_perm_S (w3[2], w3[1], selector);
      c2[1] = hc_byte_perm_S (w3[1], w3[0], selector);
      c2[0] = hc_byte_perm_S (w3[0], w2[3], selector);
      c1[3] = hc_byte_perm_S (w2[3], w2[2], selector);
      c1[2] = hc_byte_perm_S (w2[2], w2[1], selector);
      c1[1] = hc_byte_perm_S (w2[1], w2[0], selector);
      c1[0] = hc_byte_perm_S (w2[0], w1[3], selector);
      c0[3] = hc_byte_perm_S (w1[3], w1[2], selector);
      c0[2] = hc_byte_perm_S (w1[2], w1[1], selector);
      c0[1] = hc_byte_perm_S (w1[1], w1[0], selector);
      c0[0] = hc_byte_perm_S (w1[0], w0[3], selector);
      w3[3] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[2] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[1] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[0] = hc_byte_perm_S (w0[0],     0, selector);
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 13:
      c3[1] = hc_byte_perm_S (    0, w3[3], selector);
      c3[0] = hc_byte_perm_S (w3[3], w3[2], selector);
      c2[3] = hc_byte_perm_S (w3[2], w3[1], selector);
      c2[2] = hc_byte_perm_S (w3[1], w3[0], selector);
      c2[1] = hc_byte_perm_S (w3[0], w2[3], selector);
      c2[0] = hc_byte_perm_S (w2[3], w2[2], selector);
      c1[3] = hc_byte_perm_S (w2[2], w2[1], selector);
      c1[2] = hc_byte_perm_S (w2[1], w2[0], selector);
      c1[1] = hc_byte_perm_S (w2[0], w1[3], selector);
      c1[0] = hc_byte_perm_S (w1[3], w1[2], selector);
      c0[3] = hc_byte_perm_S (w1[2], w1[1], selector);
      c0[2] = hc_byte_perm_S (w1[1], w1[0], selector);
      c0[1] = hc_byte_perm_S (w1[0], w0[3], selector);
      c0[0] = hc_byte_perm_S (w0[3], w0[2], selector);
      w3[3] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[2] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[1] = hc_byte_perm_S (w0[0],     0, selector);
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 14:
      c3[2] = hc_byte_perm_S (    0, w3[3], selector);
      c3[1] = hc_byte_perm_S (w3[3], w3[2], selector);
      c3[0] = hc_byte_perm_S (w3[2], w3[1], selector);
      c2[3] = hc_byte_perm_S (w3[1], w3[0], selector);
      c2[2] = hc_byte_perm_S (w3[0], w2[3], selector);
      c2[1] = hc_byte_perm_S (w2[3], w2[2], selector);
      c2[0] = hc_byte_perm_S (w2[2], w2[1], selector);
      c1[3] = hc_byte_perm_S (w2[1], w2[0], selector);
      c1[2] = hc_byte_perm_S (w2[0], w1[3], selector);
      c1[1] = hc_byte_perm_S (w1[3], w1[2], selector);
      c1[0] = hc_byte_perm_S (w1[2], w1[1], selector);
      c0[3] = hc_byte_perm_S (w1[1], w1[0], selector);
      c0[2] = hc_byte_perm_S (w1[0], w0[3], selector);
      c0[1] = hc_byte_perm_S (w0[3], w0[2], selector);
      c0[0] = hc_byte_perm_S (w0[2], w0[1], selector);
      w3[3] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[2] = hc_byte_perm_S (w0[0],     0, selector);
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;

    case 15:
      c3[3] = hc_byte_perm_S (    0, w3[3], selector);
      c3[2] = hc_byte_perm_S (w3[3], w3[2], selector);
      c3[1] = hc_byte_perm_S (w3[2], w3[1], selector);
      c3[0] = hc_byte_perm_S (w3[1], w3[0], selector);
      c2[3] = hc_byte_perm_S (w3[0], w2[3], selector);
      c2[2] = hc_byte_perm_S (w2[3], w2[2], selector);
      c2[1] = hc_byte_perm_S (w2[2], w2[1], selector);
      c2[0] = hc_byte_perm_S (w2[1], w2[0], selector);
      c1[3] = hc_byte_perm_S (w2[0], w1[3], selector);
      c1[2] = hc_byte_perm_S (w1[3], w1[2], selector);
      c1[1] = hc_byte_perm_S (w1[2], w1[1], selector);
      c1[0] = hc_byte_perm_S (w1[1], w1[0], selector);
      c0[3] = hc_byte_perm_S (w1[0], w0[3], selector);
      c0[2] = hc_byte_perm_S (w0[3], w0[2], selector);
      c0[1] = hc_byte_perm_S (w0[2], w0[1], selector);
      c0[0] = hc_byte_perm_S (w0[1], w0[0], selector);
      w3[3] = hc_byte_perm_S (w0[0],     0, selector);
      w3[2] = 0;
      w3[1] = 0;
      w3[0] = 0;
      w2[3] = 0;
      w2[2] = 0;
      w2[1] = 0;
      w2[0] = 0;
      w1[3] = 0;
      w1[2] = 0;
      w1[1] = 0;
      w1[0] = 0;
      w0[3] = 0;
      w0[2] = 0;
      w0[1] = 0;
      w0[0] = 0;

      break;
  }
  #endif
}

__device__ void sha1_update_64 (sha1_ctx_t *ctx, u32 *w0, u32 *w1, u32 *w2, u32 *w3, const int len)
{
  const int pos = ctx->len & 63;

  ctx->len += len;

  if ((pos + len) < 64)
  {
    switch_buffer_by_offset_be_S (w0, w1, w2, w3, pos);

    ctx->w0[0] |= w0[0];
    ctx->w0[1] |= w0[1];
    ctx->w0[2] |= w0[2];
    ctx->w0[3] |= w0[3];
    ctx->w1[0] |= w1[0];
    ctx->w1[1] |= w1[1];
    ctx->w1[2] |= w1[2];
    ctx->w1[3] |= w1[3];
    ctx->w2[0] |= w2[0];
    ctx->w2[1] |= w2[1];
    ctx->w2[2] |= w2[2];
    ctx->w2[3] |= w2[3];
    ctx->w3[0] |= w3[0];
    ctx->w3[1] |= w3[1];
    ctx->w3[2] |= w3[2];
    ctx->w3[3] |= w3[3];
  }
  else
  {
    u32 c0[4] = { 0 };
    u32 c1[4] = { 0 };
    u32 c2[4] = { 0 };
    u32 c3[4] = { 0 };

    switch_buffer_by_offset_carry_be_S (w0, w1, w2, w3, c0, c1, c2, c3, pos);

    ctx->w0[0] |= w0[0];
    ctx->w0[1] |= w0[1];
    ctx->w0[2] |= w0[2];
    ctx->w0[3] |= w0[3];
    ctx->w1[0] |= w1[0];
    ctx->w1[1] |= w1[1];
    ctx->w1[2] |= w1[2];
    ctx->w1[3] |= w1[3];
    ctx->w2[0] |= w2[0];
    ctx->w2[1] |= w2[1];
    ctx->w2[2] |= w2[2];
    ctx->w2[3] |= w2[3];
    ctx->w3[0] |= w3[0];
    ctx->w3[1] |= w3[1];
    ctx->w3[2] |= w3[2];
    ctx->w3[3] |= w3[3];

    sha1_transform (ctx->w0, ctx->w1, ctx->w2, ctx->w3, ctx->h);

    ctx->w0[0] = c0[0];
    ctx->w0[1] = c0[1];
    ctx->w0[2] = c0[2];
    ctx->w0[3] = c0[3];
    ctx->w1[0] = c1[0];
    ctx->w1[1] = c1[1];
    ctx->w1[2] = c1[2];
    ctx->w1[3] = c1[3];
    ctx->w2[0] = c2[0];
    ctx->w2[1] = c2[1];
    ctx->w2[2] = c2[2];
    ctx->w2[3] = c2[3];
    ctx->w3[0] = c3[0];
    ctx->w3[1] = c3[1];
    ctx->w3[2] = c3[2];
    ctx->w3[3] = c3[3];
  }
}

__device__ void sha1_update (sha1_ctx_t *ctx, const u32 *w, const int len)
{
  u32 w0[4];
  u32 w1[4];
  u32 w2[4];
  u32 w3[4];

  int pos1;
  int pos4;

  for (pos1 = 0, pos4 = 0; pos1 < len - 64; pos1 += 64, pos4 += 16)
  {
    w0[0] = w[pos4 +  0];
    w0[1] = w[pos4 +  1];
    w0[2] = w[pos4 +  2];
    w0[3] = w[pos4 +  3];
    w1[0] = w[pos4 +  4];
    w1[1] = w[pos4 +  5];
    w1[2] = w[pos4 +  6];
    w1[3] = w[pos4 +  7];
    w2[0] = w[pos4 +  8];
    w2[1] = w[pos4 +  9];
    w2[2] = w[pos4 + 10];
    w2[3] = w[pos4 + 11];
    w3[0] = w[pos4 + 12];
    w3[1] = w[pos4 + 13];
    w3[2] = w[pos4 + 14];
    w3[3] = w[pos4 + 15];

    sha1_update_64 (ctx, w0, w1, w2, w3, 64);
  }

  w0[0] = w[pos4 +  0];
  w0[1] = w[pos4 +  1];
  w0[2] = w[pos4 +  2];
  w0[3] = w[pos4 +  3];
  w1[0] = w[pos4 +  4];
  w1[1] = w[pos4 +  5];
  w1[2] = w[pos4 +  6];
  w1[3] = w[pos4 +  7];
  w2[0] = w[pos4 +  8];
  w2[1] = w[pos4 +  9];
  w2[2] = w[pos4 + 10];
  w2[3] = w[pos4 + 11];
  w3[0] = w[pos4 + 12];
  w3[1] = w[pos4 + 13];
  w3[2] = w[pos4 + 14];
  w3[3] = w[pos4 + 15];

  sha1_update_64 (ctx, w0, w1, w2, w3, len - pos1);
}

__device__ void set_mark_1x4_S (u32 *v, const u32 offset)
{
  const u32 c = (offset & 15) / 4;
  const u32 r = 0xff << ((offset & 3) * 8);

  v[0] = (c == 0) ? r : 0;
  v[1] = (c == 1) ? r : 0;
  v[2] = (c == 2) ? r : 0;
  v[3] = (c == 3) ? r : 0;
}

__device__ void append_helper_1x4_S (u32 *r, const u32 v, const u32 *m)
{
  r[0] |= v & m[0];
  r[1] |= v & m[1];
  r[2] |= v & m[2];
  r[3] |= v & m[3];
}

__device__ void append_0x80_4x4_S (u32 *w0, u32 *w1, u32 *w2, u32 *w3, const u32 offset)
{
  u32 v[4];

  set_mark_1x4_S (v, offset);

  const u32 offset16 = offset / 16;

  append_helper_1x4_S (w0, ((offset16 == 0) ? 0x80808080 : 0), v);
  append_helper_1x4_S (w1, ((offset16 == 1) ? 0x80808080 : 0), v);
  append_helper_1x4_S (w2, ((offset16 == 2) ? 0x80808080 : 0), v);
  append_helper_1x4_S (w3, ((offset16 == 3) ? 0x80808080 : 0), v);
}

__device__ void sha1_final(sha1_ctx_t *ctx)
{
	volatile const int pos = ctx->len & 63;

	append_0x80_4x4_S(ctx->w0, ctx->w1, ctx->w2, ctx->w3, pos ^ 3);

	if (pos >= 56)
	{
		sha1_transform(ctx->w0, ctx->w1, ctx->w2, ctx->w3, ctx->h);

		ctx->w0[0] = 0;
		ctx->w0[1] = 0;
		ctx->w0[2] = 0;
		ctx->w0[3] = 0;
		ctx->w1[0] = 0;
		ctx->w1[1] = 0;
		ctx->w1[2] = 0;
		ctx->w1[3] = 0;
		ctx->w2[0] = 0;
		ctx->w2[1] = 0;
		ctx->w2[2] = 0;
		ctx->w2[3] = 0;
		ctx->w3[0] = 0;
		ctx->w3[1] = 0;
		ctx->w3[2] = 0;
		ctx->w3[3] = 0;
	}

	ctx->w3[2] = 0;
	ctx->w3[3] = ctx->len * 8;

	sha1_transform(ctx->w0, ctx->w1, ctx->w2, ctx->w3, ctx->h);
}

__device__ void SHA1(
	char hash_out[20],
	const char *str,
	int len)
{
	sha1_ctx_t ctx;
	unsigned int ii;

	sha1_init(&ctx);
	u32 input_b[] = { 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u };

	for (ii = 0; ii < len; ii += 1) {
		input_b[0] = str[ii] << 24;
		sha1_update(&ctx, input_b, 1);

		/*
		printf("hash sha1 buffer @ %d ", ii);		
		for (int i = 0; i < 16; i++)
		{
			printf("%02x", ((ctx.w0[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
		}
		for (int i = 0; i < 16; i++)
		{
			printf("%02x", ((ctx.w1[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
		}
		for (int i = 0; i < 16; i++)
		{
			printf("%02x", ((ctx.w2[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
		}
		for (int i = 0; i < 16; i++)
		{
			printf("%02x", ((ctx.w3[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
		}
		printf("\n");
		printf("hash sha1 state @ %d ", ii);
		for (int i = 0; i < 20; i++)
		{
			printf("%02x", (unsigned char)
				((ctx.h[i >> 2] >> ((3 - (i & 3)) * 8)) & 255));
		}
		printf("\n");
		*/
	}
	sha1_final(&ctx);
	for (int i = 0; i < 20; i++)
	{
		hash_out[i] = (unsigned char)
			((ctx.h[i >> 2] >> ((3 - (i & 3)) * 8)) & 255);
	}
}